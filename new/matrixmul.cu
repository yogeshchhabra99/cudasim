#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

__global__
void vecMulMatrixKernel(float* A, float* B, float* C, int n){
	int column = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	//printf("%d ",blockDim.x);
	if(row<n && column <n){
			
		float val = 0.0;
		int i;
		for(i=0;i<n;i++){
				val += A[row*n+i] * B[i*n+column];			
		}
		C[row*n+column]=val;
	}
}

__host__
void vecMulMatrix(float* A,float* B,float* C, int n){
	int size = n * n * sizeof(float);
	float *d_A, *d_B, *d_C;
	
	//Allocate device memory for A,B,C
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);	

	//copy A,B to device memory
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	//call kernal function that the calculates the product and stores it in C
	dim3 dimBlock(16,16,1);
	dim3 dimGrid(ceil(n/16.0),ceil(n/16.0),1);
	vecMulMatrixKernel<<<dimGrid,dimBlock >>>(d_A,d_B,d_C,n);		

	//copy C from devce memory
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	
	//free device memories
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

//Kernal function that runs in each thread


int main(){
	int n=256;
	int i,j;
	float A[n][n],C[n][n],B[n][n];
	for(i=0;i<n;i++){
		for(j=0;j<n;j++){
			A[i][j]=(i+j)%1024;
			B[i][j]=(i*j)%1024;
		}
	}
	
	vecMulMatrix(&A[0][0],&B[0][0],&C[0][0],n);
	for(i=0;i<n;i++){
		for(j=0;j<n;j++){
			printf("%.3f ",A[i][j]);
		}
		printf("\n");
	}
	printf("---\n");
	for(i=0;i<n;i++){
		for(j=0;j<n;j++){
			printf("%.3f ",B[i][j]);
		}
		printf("\n");
	}
	printf("---\n");
	for(i=0;i<n;i++){
		for(j=0;j<n;j++){
			printf("%.3f ",C[i][j]);
		}
		printf("\n");
	}
	
	return 0;
}




