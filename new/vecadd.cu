#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

__global__
void vecAddKernel(float* A, float* B, float* C, int n){
	//identify the index of the data to be read
	int i= threadIdx.x + blockDim.x * blockIdx.x;
	//calculate the sum and store
	if(i<n)
		C[i] = A[i] + B[i];
}

__host__
void vecAdd(float* A,float* B,float* C, int n){
	int c=ceil(n/256.0);
	int size = n * sizeof(float);
	float *d_A, *d_B, *d_C;

	//Allocate device memory for A,B,C
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);	

	//copy A,B to device memory
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	//call kernal function that the calculates sum and stores it in C
	vecAddKernel<<< ceil(n/256.0),256 >>>(d_A,d_B,d_C,n);		
	//the y and z dimensions are set to 1 by default


	//copy C from devce memory
	hipMemcpy( C,d_C, size, hipMemcpyDeviceToHost);
	
	//free device memories
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

//Kernal function that runs in each thread


int main(){
	float *A,*B,*C;
	int n=10;
	A=(float*)malloc(n*sizeof(float));
	B=(float*)malloc(n*sizeof(float));
	C=(float*)malloc(n*sizeof(float));
	int i;
	for(i=0;i<n;i++){
		A[i]=(float)i;
		B[i]=(float)2*i;	
	}
	vecAdd(A,B,C,n);
	for(i=0;i<n;i++){
		printf("%f ",C[i]);	
	}
	free(A);
	free(B);
	free(C);
	return 0;
}




