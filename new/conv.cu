#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

__global__
void vecConvKernel(float* A, float* B, float* C,int*D, int n,int m){
	//identify the index of the data to be read
	int i= threadIdx.x + blockDim.x * blockIdx.x;
	int j;
//	if(threadIdx.x==0)
//		printf("threadIdx.x is 0, blockid: %d\n",blockIdx.x);
	float val=0.0;
	int N=n+m-2;
	int start=0;
	int end=n;
	//calculate the sum and store
	if(i<=N){
		
		if(i-m>start)
			start=i-m;
		if(i<end)
			end=i;
		printf("start:%d   end:%d\n",start,end);
		for(j=start;j<=end;j++){
			val+=A[j]*B[i-j];
		}
	}
//	else{
//		printf("i:%d n:%d blockdim:%d\n",i,N,blockDim.x);		
//	}
	C[i]=val;
	D[i]=end;
}

__host__
void vecConv(float* A,float* B,float* C,int* D, int n, int m){
	int c=ceil(n/256.0);
	int size1 = n * sizeof(float);
	int size2 = m * sizeof(float);
	int size3 = (n+m-1) * sizeof(float);
	float *d_A, *d_B, *d_C;	//A of size n and B of size m
	int *d_D;
	//Allocate device memory for A,B,C
	hipMalloc((void**)&d_A, size1);
	hipMalloc((void**)&d_B, size2);
	hipMalloc((void**)&d_C, size3);	
	hipMalloc((void**)&d_D, size3);	

	//copy A,B to device memory
	hipMemcpy(d_A, A, size1, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size2, hipMemcpyHostToDevice);

	//call kernal function that the calculates sum and stores it in C

	double blocks = ceil((n+m-1)/256.0);
	printf("blocks:%lf\n",blocks);
	clock_t start,end;
	start=clock();

	//printf("n+m:%d,%f blocks:%d,%f\n",(n+m-1),(n+m-1),ceil((float)(n+m-1)/256.0),ceil((float)(n+m-1)/256.0));
	vecConvKernel<<< blocks,256 >>>(d_A,d_B,d_C,d_D,n,m);		
	hipDeviceSynchronize();
	end=clock();
	printf("time:%lf\n",(double)(end-start)/CLOCKS_PER_SEC);

	//the y and z dimensions are set to 1 by default


	//copy C from devce memory
	hipMemcpy( C,d_C, size3, hipMemcpyDeviceToHost);
	hipMemcpy( D,d_D, size3, hipMemcpyDeviceToHost);
	//free device memories
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipFree(d_D);
}

//Kernal function that runs in each thread


int main(){
	float *A,*B,*C;
	int*D;
	int n=1024*1024;
	A=(float*)malloc(n*sizeof(float));
	B=(float*)malloc(n*sizeof(float));
	C=(float*)malloc((n+n-1)*sizeof(float));
	D=(int*)malloc((n+n-1)*sizeof(int));
	int i;
	for(i=0;i<n;i++){
		A[i]=(float)i;
		B[i]=(float)2*i;	
	}
	vecConv(A,B,C,D,n,n);
//	for(i=1024*1023;i<1024*1024;i++){
//		printf("%d\n",D[i]);	
//	}
	free(A);
	free(B);
	free(C);
	return 0;
}




