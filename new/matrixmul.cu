#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<ctime>
__global__
void vecMulMatrixKernel(float* A, float* B, float* C, int n){
//	clock_t start = clock();
	int column = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	//printf("%d ",blockDim.x);
	if(row<n && column <n){
			
		float val = 0.0;
		int i;
		for(i=0;i<n;i++){
				val += A[row*n+i] * B[i*n+column];			
		}
		C[row*n+column]=val;
	}
//	clock_t end =clock();
//	printf("time:%lld\n",(int)(end-start));

}

__host__
void vecMulMatrix(float* A,float* B,float* C, int n){
	int size = n * n * sizeof(float);
	float *d_A, *d_B, *d_C;

	printf("mem:%dKb\n",3*size/1024);
	
	//Allocate device memory for A,B,C
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);	

	//copy A,B to device memory
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	//call kernal function that the calculates the product and stores it in C
	dim3 dimBlock(16,16,1);
	dim3 dimGrid(ceil(n/16.0),ceil(n/16.0),1);
	
	time_t t1=time(NULL);
//	for(int i=0;i<33;i++)		//315 when n=640   33 when n=1024
	vecMulMatrixKernel<<<dimGrid,dimBlock >>>(d_A,d_B,d_C,n);		
	hipDeviceSynchronize();
	time_t t2=time(NULL);
	printf("%d\n",t2-t1);
	//copy C from devce memory
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	
	//free device memories
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

//Kernal function that runs in each thread


int main(){
	int n=1024;//640;
	int i,j;
//	float A[n][n],C[n][n],B[n][n];
//	for(i=0;i<n;i++){
//		for(j=0;j<n;j++){
//			A[i][j]=(i+j)%1024;
//			B[i][j]=(i*j)%1024;
//		}
//	}
	
	float **A=(float**)malloc(1024*sizeof(float*));
	float **B=(float**)malloc(1024*sizeof(float*));
	float **C=(float**)malloc(1024*sizeof(float*));
	for(int i=0;i<1024;i++){
		A[i]=(float*)malloc(1024*sizeof(float));
		B[i]=(float*)malloc(1024*sizeof(float));
		C[i]=(float*)malloc(1024*sizeof(float));
	}
	vecMulMatrix(&A[0][0],&B[0][0],&C[0][0],n);
//	vecMulMatrix(A,B,C,n);
//	for(i=0;i<n;i++){
//		for(j=0;j<n;j++){
//			printf("%.3f ",A[i][j]);
//		}
//		printf("\n");
//	}
//	printf("---\n");
//	for(i=0;i<n;i++){
//		for(j=0;j<n;j++){
//			printf("%.3f ",B[i][j]);
//		}
//		printf("\n");
//	}
//	printf("---\n");
//	for(i=0;i<n;i++){
//		for(j=0;j<n;j++){
//			printf("%.3f ",C[i][j]);
//		}
//		printf("\n");
//	}
	
	return 0;
}




