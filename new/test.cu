#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<ctime>

__global__
void vecMulMatrixKernel(float* A, float* B, float* C, int n){
//	clock_t start = clock();
	int column = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	//printf("%d ",blockDim.x);
	if(row<n && column <n){
			
		float val = 0.0;
		int i;
		for(i=0;i<n;i++){
				val += A[row*n+i] * B[i*n+column];			
		}
		C[row*n+column]=val;
	}
//	clock_t end =clock();
//	printf("time:%lld\n",(int)(end-start));

}

int main(){}