#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<time.h>

__global__
void mKernal(int *A){
	int i=0;
	__shared__ int arr[10];
	arr[0]=6;
	//while(i<1000000000){		//2147483647
		i=i*1;	
		i=i+1;
		
		//printf(";");
	//}
	//*A=i;
	*A=arr[0];
}

__host__
int main(){
	time_t t1,t2;
	
	
	long n=pow(2,34);
	long nb=n/256;
	int* dA;
	hipMalloc((void**)&dA, 4);
	//int A;
	
	//printf("n: %ld nb: %ld A: %d\n",n,nb,A);
	dim3 dim(nb,4,1);
	t1=time(NULL);
	mKernal<<<dim,256>>>(dA);
	//mKernal<<<1,1>>>(&A);
	hipDeviceSynchronize();
	t2=time(NULL);
	hipMemcpy( &A,dA, 4, hipMemcpyDeviceToHost);	
	printf("Time:%ld   A: %d\n",t2-t1,A);
	
}
