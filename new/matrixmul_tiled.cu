	//
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<ctime>
#include<time.h>
#define TILEWIDTH 32
#define TILE_WIDTH 32

__global__
void vecMulMatrixKernel(float* A, float* B, float* C, int n){
	//each block loads the corresponding row of blocks of A matrix and column of blocks of B matrix, one block at a time and then clculates the product for that part then product of a  the parts is added.
	// each thread loads 2 elements one from A and one from B in each phase
	// there are total gridDim.x phases
	// the element loaded is the element at the same position as this thread but in a different block
	//if run more thread than max then not run
	int tx=threadIdx.x; int ty=threadIdx.y;
	int bx=blockIdx.x; int by=blockIdx.y;
	
	int row=by*blockDim.y+ty;
	int col=bx*blockDim.x+tx;
	
	__shared__ float Ads[TILEWIDTH][TILEWIDTH];
	__shared__ float Bds[TILEWIDTH][TILEWIDTH];
	
	if(row<n && col <n){
		int i; float val=0.0;
		for(i=0;i<gridDim.x-1;i++){
			Ads[ty][tx] = A[ row*n + i*TILEWIDTH + tx];
			Bds[ty][tx] = B[ (i*TILEWIDTH + ty)*n + col];
			__syncthreads();
			for(int k=0;k<TILEWIDTH;k++){
				val+= Ads[ty][k]*Bds[tx][k];	
			}
			__syncthreads();
			
		}

		if(i*TILEWIDTH + tx <n )		//if n was a multiple of blockDim then this was not required
			Ads[ty][tx] = A[ row*n + i*TILEWIDTH + tx];
		if(i*TILEWIDTH + ty <n )
			Bds[ty][tx] = B[ (i*TILEWIDTH + ty)*n + col];
		__syncthreads();
		int m =n%TILEWIDTH;
		if(m==0)
			m=TILEWIDTH;
		for(int k=0;k<m;k++){//printf("add");
			val+= Ads[ty][k]*Bds[tx][k];	
		}
		__syncthreads();
		C[row*n + col]= val;
	}
}

__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width)
{  
__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];  
int bx = blockIdx.x;
int by = blockIdx.y;
int tx = threadIdx.x;
int ty = threadIdx.y;
// Identify the row and column of the Pd element to work on
int Row = by * TILE_WIDTH + ty;
int Col = bx * TILE_WIDTH + tx;
float Pvalue = 0;
// Loop over the Md and Nd tiles required to compute the Pd element
for (int m = 0; m < Width/TILE_WIDTH; ++m) {
// Collaborative loading of Md and Nd tiles into shared memory
Mds[ty][tx] = Md[Row*Width + (m*TILE_WIDTH + tx)];
Nds[ty][tx] = Nd[Col + (m*TILE_WIDTH + ty)*Width];
__syncthreads();
for (int k = 0; k < TILE_WIDTH; ++k)
Pvalue += Mds[ty][k] * Nds[k][tx];
__syncthreads();
}
Pd[Row*Width+Col] = Pvalue;
}


int min2Power(int x){
	int res=1;
	while(res<x){
		res*=2;	
	}
	return res/2;
}

__host__
void vecMulMatrix(float* A,float* B,float* C, int n){
	int size = n * n * sizeof(float);
	float *d_A, *d_B, *d_C;
	
	//Allocate device memory for A,B,C
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	//copy A,B to device memory
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	
	//call kernal function that the calculates the product and stores it in C
	dim3 dimBlock(TILEWIDTH,TILEWIDTH,1);
	dim3 dimGrid(ceil(n/(float)TILEWIDTH),ceil(n/(float)TILEWIDTH),1);

//	time_t t1=time(NULL);
//	for(int i=0;i<270;i++)
//	vecMulMatrixKernel<<<dimGrid,dimBlock >>>(d_A,d_B,d_C,n);		

	clock_t start,end;
	start=clock();
	MatrixMulKernel<<<dimGrid,dimBlock>>>(d_A,d_B,d_C,n);
	hipDeviceSynchronize();
	end=clock();
	printf("time:%lf\n",(double)(end-start)/CLOCKS_PER_SEC);

//	time_t t2=time(NULL);
//	printf("%d\n",t2-t1);

	//copy C from devce memory
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	
	//free device memories
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

//Kernal function that runs in each thread


int main(){
	int maxThreads=640;
	
		int n=1024*2;//640;
	int i,j;
//	float A[n][n],C[n][n],B[n][n];
//	for(i=0;i<n;i++){
//		for(j=0;j<n;j++){
//			A[i][j]=(i+j)%1024;
//			B[i][j]=(i*j)%1024;
//		}
//	}
	
	float **A=(float**)malloc(1024*sizeof(float*));
	float **B=(float**)malloc(1024*sizeof(float*));
	float **C=(float**)malloc(1024*sizeof(float*));
	for(int i=0;i<1024;i++){
		A[i]=(float*)malloc(1024*sizeof(float));
		B[i]=(float*)malloc(1024*sizeof(float));
		C[i]=(float*)malloc(1024*sizeof(float));
	}
	vecMulMatrix(&A[0][0],&B[0][0],&C[0][0],n);

//for(i=0;i<n;i++){
//		for(j=0;j<n;j++){
	//		printf("%.3f ",A[i][j]);
//		}
	//	printf("\n");
//	}
//	printf("---\n");
//	for(i=0;i<n;i++){
//		for(j=0;j<n;j++){
	//		printf("%.3f ",B[i][j]);
//		}
	//	printf("\n");
//	}
//	printf("---\n");
//	for(i=0;i<n;i++){
//		for(j=0;j<n;j++){
	//		printf("%.3f ",C[i][j]);
//		}
//		printf("\n");
//	}
	
	return 0;
}




