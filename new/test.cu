#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

__global__
void vecMulMatrixKernel(float* A, float* B, float* C, float* D, int n){
	int column = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	printf("%d ",blockDim.x);
	if(row<n && column <n){
			
		float val = 0.0;
		int i,j;
		for(i=0;i<n;i++){
			for( j=0;j<n;j++){
					val += A[row*n+i] * B[j*n+column];
			}
			 val += A[row*n+i] * C[i*n+column];
		}
		C[row*n+column]=val;
	}
}

__host__
void vecMulMatrix(float* A,float* B,float* C, float* D, int n){
	int size = n * n * sizeof(float);
	float *d_A, *d_B, *d_C, *d_D;
	
	//Allocate device memory for A,B,C
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);
	hipMalloc((void**)&d_D, size);	

	//copy A,B to device memory
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	hipMemcpy(d_C, B, size, hipMemcpyHostToDevice);

	//call kernal function that the calculates the product and stores it in C
	dim3 dimBlock(16,16,1);
	dim3 dimGrid(ceil(n/16.0),ceil(n/16.0),1);
	vecMulMatrixKernel<<<dimGrid,dimBlock >>>(d_A,d_B,d_C,d_D,n);		

	//copy C from devce memory
	hipMemcpy(D, d_D, size, hipMemcpyDeviceToHost);
	
	//free device memories
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipFree(d_D);
}

//Kernal function that runs in each thread


int main(){
	int n=256;
	int i,j;
	float A[n][n],C[n][n],B[n][n],D[n][n];
	for(i=0;i<n;i++){
		for(j=0;j<n;j++){
			A[i][j]=(i+j)%1024;
			B[i][j]=(i*j)%1024;
			C[i][j]=(i%j)%1024;
		}
	}
	
	vecMulMatrix(&A[0][0],&B[0][0],&C[0][0],&D[0][0],n);
	for(i=0;i<n;i++){
		for(j=0;j<n;j++){
			printf("%.3f ",A[i][j]);
		}
		printf("\n");
	}
	printf("---\n");
	for(i=0;i<n;i++){
		for(j=0;j<n;j++){
			printf("%.3f ",B[i][j]);
		}
		printf("\n");
	}
	printf("---\n");
	for(i=0;i<n;i++){
		for(j=0;j<n;j++){
			printf("%.3f ",D[i][j]);
		}
		printf("\n");
	}
	
	return 0;
}




